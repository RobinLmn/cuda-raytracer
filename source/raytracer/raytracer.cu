#include "hip/hip_runtime.h"
#include "raytracer.hpp"

#include "core/log.hpp"

#include "raytracer/ray.hpp"
#include "raytracer/hit_info.hpp"

namespace rAI
{
    __device__ hit_info ray_sphere_intersection(const ray& r, const sphere& s)
    {
        const glm::vec3 oc = r.origin - s.center;
        const float a = glm::dot(r.direction, r.direction);
        const float b = 2.0f * glm::dot(oc, r.direction);
        const float c = glm::dot(oc, oc) - s.radius * s.radius;

        const float discriminant = b * b - 4.0f * a * c;
        if (discriminant < 0.0f)
            return hit_info{ false, 0.0f, glm::vec3(0.0f), glm::vec3(0.0f) };

        float distance = (-b - sqrt(discriminant)) / (2.0f * a);
        if (distance < 0.0f)
            return hit_info{ false, 0.0f, glm::vec3(0.0f), glm::vec3(0.0f) };

        const glm::vec3 point = r.origin + distance * r.direction;
        const glm::vec3 normal = glm::normalize(point - s.center);
        
        return hit_info{ true, distance, point, normal };
    }

    __device__ glm::vec3 trace(const scene& scene, const ray& ray)
    {
        for (int i = 0; i < scene.spheres_count; i++)
        {
            hit_info hit = ray_sphere_intersection(ray, scene.spheres[i]);

            if (hit.did_hit)
                return hit.normal;
        }

        const float a = 0.5f * (ray.direction.y + 1.0f);
        return (1.0f - a) * glm::vec3{ 1.0f, 1.0f, 1.0f } + a * glm::vec3{ 0.5f, 0.7f, 1.0f };
    }

    __global__ void write_to_texture(hipSurfaceObject_t surface, int width, int height, const rendering_context rendering_context, const scene scene)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        
        if (y >= height || x >= width)
            return;

        const glm::vec2 uv = glm::vec2{ (float)x / (float)width, 1.0f - (float)y / (float)height } * 2.0f - 1.0f;
        const glm::vec4 target = rendering_context.inverse_projection_matrix * glm::vec4{ uv, 1.0f, 1.0f };

        const glm::vec3 ray_origin = rendering_context.camera_position;
        const glm::vec3 ray_direction = glm::vec3{ rendering_context.inverse_view_matrix * glm::vec4{ glm::normalize(glm::vec3{ target } / target.w), 0.0f } };

        ray ray{ ray_origin, ray_direction };
        
        const glm::vec3 color = trace(scene, ray);

        uchar4 color_u = make_uchar4(color.r * 255, color.g * 255, color.b * 255, 255);
        surf2Dwrite(color_u, surface, x * sizeof(uchar4), y);
    }

    raytracer::raytracer(const int width, const int height)
        : width{ width }
        , height{ height }
        , cuda_texture_resource{ nullptr }
        , cuda_array{ nullptr }
        , cuda_surface_write{ 0 }
        , render_texture{ width, height }
    {
        hipGraphicsGLRegisterImage(&cuda_texture_resource, render_texture.get_id(), GL_TEXTURE_2D,  hipGraphicsRegisterFlagsSurfaceLoadStore);
        hipGraphicsMapResources(1, &cuda_texture_resource, 0);
        hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_texture_resource, 0, 0);
        hipGraphicsUnmapResources(1, &cuda_texture_resource, 0);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuda_array;
        hipCreateSurfaceObject(&cuda_surface_write, &resDesc);
    }
    
    raytracer::~raytracer()
    {
        hipDestroySurfaceObject(cuda_surface_write);
        hipFreeArray(cuda_array);
        hipGraphicsUnregisterResource(cuda_texture_resource);
    }

    void raytracer::render(const rendering_context& rendering_context, const scene& scene)
    {
        const int thread_x = 16;
        const int thread_y = 16;

        dim3 blocks((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
        dim3 threads(thread_x, thread_y);

        write_to_texture<<<blocks, threads>>>(cuda_surface_write, width, height, rendering_context, scene);

#ifdef DEBUG
        [[maybe_unused]] hipError_t error = hipGetLastError();
        ASSERT(!error, "[CUDA] {} : {}", hipGetErrorName(error), hipGetErrorString(error));
#endif
            
        hipDeviceSynchronize();
    }

    unsigned int raytracer::get_render_texture() const
    {
        return render_texture.get_id();
    }
}
