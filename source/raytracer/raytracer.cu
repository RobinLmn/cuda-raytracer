#include "hip/hip_runtime.h"
#include "raytracer.hpp"

#include "core/log.hpp"

#include "raytracer/random.cuh"
#include "raytracer/intersection.cuh"

#include <hiprand/hiprand_kernel.h>

namespace rAI
{
    __device__ glm::vec3 get_sky_light(const sky_box& sky, const ray& ray)
    {
        const float horizon_to_zenith_gradient_t = powf(glm::smoothstep(0.0f, 0.4f, ray.direction.y), 0.35f);
        const glm::vec3 horizon_to_zenith_gradient = glm::mix(sky.horizon_color, sky.zenith_color, horizon_to_zenith_gradient_t);

        const float ground_to_sky_gradient_t = glm::smoothstep(-0.01f, 0.0f, ray.direction.y);
        const glm::vec3 ground_to_sky_gradient = glm::mix(sky.ground_color, horizon_to_zenith_gradient, ground_to_sky_gradient_t);

        const float sun = powf(max(0.0f, glm::dot(ray.direction, glm::normalize(sky.sun_direction))), sky.sun_focus) * sky.sun_intensity;
        
        return ground_to_sky_gradient + sun;
    }

    __device__ hit_info get_closest_hit(const scene& scene, const ray& ray)
    {
        hit_info closest_hit{ false, FLT_MAX, glm::vec3{ 0.0f }, glm::vec3{ 0.0f } };

        for (int i = 0; i < scene.spheres_count; i++)
        {
            hit_info hit = ray_sphere_intersection(ray, scene.spheres[i]);

            if (hit.did_hit && hit.distance < closest_hit.distance)
                closest_hit = hit;
        }

        return closest_hit;
    }

    __device__ glm::vec3 trace(const scene& scene, const ray& starting_ray, hiprandState& random_state, const int max_bounces, const sky_box& sky_box)
    {
        glm::vec3 incoming_light{ 0.0f };
        glm::vec3 ray_color{ 1.0f };
        ray ray = starting_ray;

        for (int i = 0; i < max_bounces; i++)
        {
            const hit_info& closest_hit = get_closest_hit(scene, ray);
            if (closest_hit.did_hit)
            {
                ray.origin = closest_hit.point;
                ray.direction = random_hemisphere_direction(closest_hit.normal, random_state);

                incoming_light += closest_hit.material.emission_strength * closest_hit.material.emission_color * ray_color;
                ray_color *= closest_hit.material.color;
            }
            else
            {
                incoming_light += get_sky_light(sky_box, ray) * ray_color;
                break;
            }
        }

        return incoming_light;
    }

    __global__ void write_to_texture(hipSurfaceObject_t surface, int width, int height, const rendering_context rendering_context, const scene scene)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        
        if (y >= height || x >= width)
            return;

        const glm::vec2 uv = glm::vec2{ (float)x / (float)width, 1.0f - (float)y / (float)height } * 2.0f - 1.0f;
        const glm::vec4 target = rendering_context.inverse_projection_matrix * glm::vec4{ uv, 1.0f, 1.0f };
        
        hiprandState random_state;
        hiprand_init(y + width * x, 0, 0, &random_state);
        
        glm::vec3 incoming_light = glm::vec3{ 0.0f };

        for (int i = 0; i < rendering_context.rays_per_pixel; i++)
        {
            const glm::vec3 ray_origin = rendering_context.camera_position;
            const glm::vec3 ray_direction = glm::vec3{ rendering_context.inverse_view_matrix * glm::vec4{ glm::normalize(glm::vec3{ target } / target.w), 0.0f } };

            ray ray{ ray_origin, ray_direction };

            incoming_light += trace(scene, ray, random_state, rendering_context.max_bounces, rendering_context.sky_box);
        }

        incoming_light /= static_cast<float>(rendering_context.rays_per_pixel);

        uchar4 color_u = make_uchar4(incoming_light.r * 255, incoming_light.g * 255, incoming_light.b * 255, 255);
        surf2Dwrite(color_u, surface, x * sizeof(uchar4), y);
    }

    __host__ raytracer::raytracer(const int width, const int height)
        : width{ width }
        , height{ height }
        , render_texture{ width, height }
    {
    }

    __host__ void raytracer::render(const rendering_context& rendering_context, const scene& scene)
    {
        const int thread_x = 16;
        const int thread_y = 16;

        dim3 blocks((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
        dim3 threads(thread_x, thread_y);

        write_to_texture<<<blocks, threads>>>(render_texture.get_surface_write(), width, height, rendering_context, scene);

        hipDeviceSynchronize();
    }

    __host__ unsigned int raytracer::get_render_texture() const
    {
        return render_texture.get_id();
    }
}
