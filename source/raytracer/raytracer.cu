#include "hip/hip_runtime.h"
#include "raytracer.hpp"

namespace rAI
{
    __global__ void write_to_texture(hipSurfaceObject_t surface, int width, int height)
    {
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (y >= height)
            return;

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x >= width)
            return;
            
        surf2Dwrite(make_uchar4(255, 0, 0, 255), surface, x * sizeof(uchar4), y);
    }

    raytracer::raytracer(const int width, const int height)
        : width{ width }
        , height{ height }
        , cuda_texture_resource{ nullptr }
        , cuda_array{ nullptr }
        , cuda_surface_write{ 0 }
        , render_texture{ width, height }
    {
        hipGraphicsGLRegisterImage(&cuda_texture_resource, render_texture.get_id(), GL_TEXTURE_2D,  hipGraphicsRegisterFlagsSurfaceLoadStore);
        hipGraphicsMapResources(1, &cuda_texture_resource, 0);
        hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_texture_resource, 0, 0);
        hipGraphicsUnmapResources(1, &cuda_texture_resource, 0);
    }
    
    raytracer::~raytracer()
    {
        hipDestroySurfaceObject(cuda_surface_write);
        hipFreeArray(cuda_array);
        hipGraphicsUnregisterResource(cuda_texture_resource);
    }

    void raytracer::render()
    {
        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = cuda_array;
        
        hipCreateSurfaceObject(&cuda_surface_write, &resDesc);
        
        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
        
        write_to_texture<<<gridDim, blockDim>>>(cuda_surface_write, width, height);
    }

    unsigned int raytracer::get_render_texture() const
    {
        return render_texture.get_id();
    }
}
