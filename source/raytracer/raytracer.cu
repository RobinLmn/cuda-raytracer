#include "hip/hip_runtime.h"
#include "raytracer.hpp"

#include "raytracer/random.cuh"
#include "raytracer/intersection.cuh"
#include "raytracer/cuda_utils.cuh"

#include <hiprand/hiprand_kernel.h>

namespace rAI
{
    __device__ glm::vec3 get_sky_light(const sky_box& sky, const ray& ray)
    {
        if (sky.is_hidden)
            return glm::vec3{ 0.f };
            
        const float horizon_to_zenith_gradient_t = powf(glm::smoothstep(0.0f, 0.4f, ray.direction.y), 0.35f);
        const glm::vec3 horizon_to_zenith_gradient = glm::mix(sky.horizon_color, sky.zenith_color, horizon_to_zenith_gradient_t);

        const float ground_to_sky_gradient_t = glm::smoothstep(-0.01f, 0.0f, ray.direction.y);
        const glm::vec3 ground_to_sky_gradient = glm::mix(sky.ground_color, horizon_to_zenith_gradient, ground_to_sky_gradient_t);

        const float sun = powf(max(0.0f, glm::dot(ray.direction, glm::normalize(sky.sun_direction))), sky.sun_focus) * sky.sun_intensity;
        
        return ground_to_sky_gradient + sun;
    }

    __device__ hit_info get_closest_hit(const scene& scene, const ray& ray)
    {
        hit_info closest_hit{ false, FLT_MAX, glm::vec3{ 0.0f }, glm::vec3{ 0.0f } };

        for (int sphere_index = 0; sphere_index < scene.spheres_count; sphere_index++)
        {
            const sphere& sphere = scene.spheres[sphere_index];
            
            const hit_info& hit = ray_sphere_intersection(ray, sphere);
            if (hit.did_hit && hit.distance < closest_hit.distance)
            {
                closest_hit = hit;
                closest_hit.material = sphere.material;
            }
        }

        for (int mesh_index = 0; mesh_index < scene.meshes_count; mesh_index++)
        {
            const mesh_info& mesh_info = scene.meshes_info[mesh_index];

            const hit_info& bounding_box_hit = ray_aabb_intersection(ray, mesh_info.bounding_box);
            if (!bounding_box_hit.did_hit)
                continue;

            for (int triangle_index = 0; triangle_index < mesh_info.triangle_count; triangle_index++)
            {
                const triangle& triangle = scene.triangles[mesh_info.triangle_start + triangle_index];
                const hit_info& hit = ray_triangle_intersection(ray, triangle);

                if (hit.did_hit && hit.distance < closest_hit.distance)
                {
                    closest_hit = hit;
                    closest_hit.material = mesh_info.material;
                }
            }
        }

        return closest_hit;
    }

    __device__ glm::vec3 trace(const scene& scene, const ray& starting_ray, hiprandState& random_state, const int max_bounces, const sky_box& sky_box)
    {
        glm::vec3 incoming_light{ 0.0f };
        glm::vec3 ray_color{ 1.0f };
        ray ray = starting_ray;

        for (int i = 0; i < max_bounces; i++)
        {
            const hit_info& closest_hit = get_closest_hit(scene, ray);
            if (closest_hit.did_hit)
            {
                const glm::vec3 diffuse = glm::normalize(closest_hit.normal + random_direction(random_state));
                const glm::vec3 specular = glm::reflect(ray.direction, closest_hit.normal);

                const bool specular_bounce = closest_hit.material.specular_probability >= random_float(random_state);

                ray.origin = closest_hit.point;
                ray.direction = glm::normalize(glm::mix(diffuse, specular, closest_hit.material.smoothness * specular_bounce));

                incoming_light += closest_hit.material.emission_strength * closest_hit.material.emission_color * ray_color;
                ray_color *= specular_bounce ? closest_hit.material.specular_color : closest_hit.material.color;

                float p = max(ray_color.r, max(ray_color.g, ray_color.b));
                if (random_float(random_state) >= p)
                    break;

                ray_color *= 1.0f / p; 
            }
            else
            {
                incoming_light += get_sky_light(sky_box, ray) * ray_color;
                break;
            }
        }

        return incoming_light;
    }

    __global__ void write_to_texture(hipSurfaceObject_t output_surface, hipSurfaceObject_t accumulation_surface, int width, int height, const rendering_context rendering_context, const scene scene, const int frame_index, const bool should_accumulate)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        if (y >= height || x >= width)
            return;

        const glm::vec2 uv = glm::vec2{ (float)x / (float)width, 1.0f - (float)y / (float)height } * 2.0f - 1.0f;
        const glm::vec4 target = rendering_context.inverse_projection_matrix * glm::vec4{ uv, 1.0f, 1.0f };

        hiprandState random_state;
        hiprand_init(y + width * x + frame_index * 719393, 0, 0, &random_state);
        
        glm::vec3 incoming_light = glm::vec3{ 0.0f };

        for (int i = 0; i < rendering_context.rays_per_pixel; i++)
        {
            const glm::vec3 direction = glm::vec3{ rendering_context.inverse_view_matrix * glm::vec4{ glm::normalize(glm::vec3{ target } / target.w), 0.0f } };
            const glm::vec3 right = glm::normalize(glm::cross(direction, glm::vec3(0.0f, 1.0f, 0.0f)));
            const glm::vec3 up = glm::normalize(glm::cross(right, direction));

            const glm::vec3 focal_point = rendering_context.camera_position + direction * rendering_context.focus_distance;

            const glm::vec2 jitter = random_point_in_circle(random_state) * rendering_context.diverge_strength / static_cast<float>(width);
            const glm::vec3 jittered_focal_point = focal_point + right * jitter.x + up * jitter.y;
            const glm::vec2 defocus_jitter = random_point_in_circle(random_state) * rendering_context.defocus_strength / static_cast<float>(width);

            const glm::vec3 ray_origin = rendering_context.camera_position + right * defocus_jitter.x + up * defocus_jitter.y;
            const glm::vec3 ray_direction = glm::normalize(jittered_focal_point - ray_origin);

            ray ray{ ray_origin, ray_direction };
            incoming_light += trace(scene, ray, random_state, rendering_context.max_bounces, rendering_context.sky_box);
        }

        incoming_light /= static_cast<float>(rendering_context.rays_per_pixel);

        const float4 new_color = make_float4(incoming_light.r, incoming_light.g, incoming_light.b, 1.f);

        if (!should_accumulate)
        {
            const uchar4 new_color_u = make_uchar4(new_color.x * 255, new_color.y * 255, new_color.z * 255, new_color.w * 255);
            surf2Dwrite(new_color_u, output_surface, x * sizeof(uchar4), y);

            return;
        }

        float4 previous_color;
        surf2Dread(&previous_color, accumulation_surface, x * sizeof(float4), y);
        
        const float4 accumulated_color = previous_color + new_color;
        surf2Dwrite(accumulated_color, accumulation_surface, x * sizeof(float4), y);

        const float4 average_color = accumulated_color / (frame_index + 1);

        const uchar4 average_color_u = make_uchar4(average_color.x * 255, average_color.y * 255, average_color.z * 255, average_color.w * 255);
        surf2Dwrite(average_color_u, output_surface, x * sizeof(uchar4), y);
    }

    __global__ void reset_accumulation_surface(hipSurfaceObject_t accumulation_surface, int width, int height)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        
        if (y >= height || x >= width)
            return;

        const float4 clear_color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        surf2Dwrite(clear_color, accumulation_surface, x * sizeof(float4), y);
    }

    __host__ raytracer::raytracer(const int width, const int height)
        : width{ width }
        , height{ height }
        , render_texture{ width, height }
        , accumulation_texture{ width, height, hipCreateChannelDesc<float4>() }
        , frame_index{ 0 }
    {
    }

    __host__ void raytracer::reset_accumulation()
    {
        const int thread_x = 16;
        const int thread_y = 16;

        dim3 blocks((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
        dim3 threads(thread_x, thread_y);

        reset_accumulation_surface<<<blocks, threads>>>(accumulation_texture.get_surface(), width, height);
        
        hipDeviceSynchronize();

        frame_index = 0;
    }

    __host__ void raytracer::render(const rendering_context& rendering_context, const scene& scene, const bool should_accumulate)
    {
        const int thread_x = 16;
        const int thread_y = 16;

        dim3 blocks((width + thread_x - 1) / thread_x, (height + thread_y - 1) / thread_y);
        dim3 threads(thread_x, thread_y);

        write_to_texture<<<blocks, threads>>>(render_texture.get_surface(), accumulation_texture.get_surface(), width, height, rendering_context, scene, frame_index, should_accumulate);

        hipDeviceSynchronize();

        if (should_accumulate)
            frame_index++;
    }

    __host__ const texture& raytracer::get_render_texture() const
    {
        return render_texture;
    }
}
